
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void vectorAddition(const float* A, const float* B, float* C, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    //initialisations
    const int N = 1000;
    const size_t arr_size = N*sizeof(float);
    float *A = new float[N];
    float *B = new float[N];
    float *C = new float[N];
    float *d_a, *d_b,*d_c;
    
    // allocate memory on GPU
    hipMalloc(&d_a, arr_size);
    hipMalloc(&d_b, arr_size);
    hipMalloc(&d_c, arr_size);

    //copy cpu vectors to gpu
    hipMemcpy(d_a, A, arr_size,hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, arr_size,hipMemcpyHostToDevice);

    // threads, blocks and function call
    int blocksize=256;
    int gridsize = (N + blocksize - 1) / blocksize;
    vectorAddition<<<gridsize,blocksize>>>(d_a, d_b, d_c, N);

    //copy gpu vectors to cpu
    hipMemcpy(C, d_c, arr_size, hipMemcpyDeviceToHost);

    //freeup the memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] A;
    delete[] B;
    delete[] C;
}